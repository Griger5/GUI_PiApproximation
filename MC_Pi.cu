#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

const int numOfThreads = 1024;

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__
void piApproximation(unsigned long long *inside, int N) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;
    __shared__ unsigned long long counter[numOfThreads];
    double x, y;

    hiprandState_t rng;
	hiprand_init(clock64(), index, 0, &rng);

    for (int i=index; i<N; i+=stride) {
        x = hiprand_uniform_double(&rng);
        y = hiprand_uniform_double(&rng);
        if (x*x+y*y < 1) {
            counter[threadIdx.x] += 1;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        inside[blockIdx.x] = 0;
        for (int i = 0; i<numOfThreads; i++) {
            inside[blockIdx.x] += counter[i];
        }
    }
}

int main() {
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    int N = 100000000;
    unsigned long long *inside;
    size_t size = sizeof(unsigned long long);

    hipMallocManaged(&inside, size);
    hipMemPrefetchAsync(inside, size, deviceId);

    int threadsPerBlock = numOfThreads;
    int blocksPerGrid = numberOfSMs;

    piApproximation<<<blocksPerGrid, threadsPerBlock>>>(inside, N);

    checkCuda(hipDeviceSynchronize());

    unsigned long long result = 0;

    for (int i=0; i<blocksPerGrid; i++) {
        result += inside[i];
    }

    hipFree(inside);

    printf("%f", 4*(double)result/N);

    return 0;
}
